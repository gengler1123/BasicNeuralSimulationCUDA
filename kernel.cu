#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <vector>
#include <ctime>
#include <iostream>

__global__ void NeuronTimestep(
	int numNeur,
	int numExcit,
	float *d_v,
	float *d_u,
	float *d_I,
	bool *d_cf,
	float *d_driven)
{

	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < numExcit)
	{
		d_cf[i] = false;
		for (int dt = 0; dt < 4; dt++)
		{
			float dv = (0.7 * (d_v[i] + 60)*(d_v[i] + 40) - d_u[i] + d_I[i] + d_driven[i]) / 100;
			float du = (0.03 * (-2*(d_v[i] + 60) - d_u[i]));
			d_v[i] += 0.25*dv;
			d_u[i] += 0.25*du;

			if (d_v[i] > 35)
			{
				d_cf[i] = true;
				d_v[i] = -50;
				d_u[i] += 100;
				break;
			}
			d_I[i] = 0;
		}


	}
	else if (i < numNeur)
	{

		d_cf[i] = false;
		for (int dt = 0; dt < 4; dt++)
		{
			float dv = (1.2 * (d_v[i] + 75)*(d_v[i] + 45) - d_u[i] + d_I[i] + d_driven[i]) / 150;
			float du = (0.01 * (5 * (d_v[i] + 75) - d_u[i]));
			d_v[i] += 0.25*dv;
			d_u[i] += 0.25*du;

			if (d_v[i] > 50)
			{
				d_cf[i] = true;
				d_v[i] = -56;
				d_u[i] += 130;
				break;
			}
		}

		d_I[i] = 0;
	}

}


__global__ void CommunicationPhase(
	int numEdge,
	bool *d_cf,
	int *d_source,
	int *d_target,
	float *d_weight,
	float *d_I)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < numEdge)
	{
		if (d_cf[d_source[i]])
		{
			atomicAdd(&d_I[d_target[i]], d_weight[i]);
		}
	}
}


int main()
{
	int numNeurons = 1000;
	int numExcit = 800;
	int T = 2000;
	int equilizationTime = 100;
	int transientTime = 300;

	/* CUDA Parameters */
	int numThreads = 512;

	/* Neurons */
	float *h_v, *d_v, *h_u, *d_u, *h_I, *d_I, *h_driven, *d_driven;
	bool *d_cf, *h_cf;

	h_v = new float[numNeurons];
	h_u = new float[numNeurons];
	h_I = new float[numNeurons];
	h_cf = new bool[numNeurons];
	h_driven = new float[numNeurons];

	bool **SpikeTrainYard = new bool*[T];
	float **VoltageTrace = new float *[T];
	for (int i = 0; i < numNeurons; i++)
	{
		h_v[i] = -60;
		h_u[i] = 0;
		h_I[i] = 0;
		h_cf[i] = false;
		if (i < 100)
		{
			h_driven[i] = 75;
		}
		else
		{
			h_driven[i] = 0;
		}
	}

	for (int t = 0; t < T; t++)
	{
		SpikeTrainYard[t] = new bool[numNeurons];
		VoltageTrace[t] = new float[numNeurons];
	}


	/* Edges */

	std::vector<int> h_source; int *d_source;
	std::vector<int> h_target; int *d_target;
	std::vector<float> h_weight; float *d_weight;
	
	std::mt19937 rd(time(NULL));
	std::uniform_real_distribution<float> dist(0.0, 1.0);

	for (int n = 0; n < numNeurons; n++)
	{
		for (int m = 0; m < numNeurons; m++)
		{
			if (n != m)
			{

				if (dist(rd) < .2)
				{
					h_source.push_back(n);
					h_target.push_back(m);
					if (n < numExcit)
					{
						h_weight.push_back(dist(rd) * 300);
					}
					else
					{
						h_weight.push_back(dist(rd) * -400);
					}
				}

			}
		}
	}

	int numEdges = h_source.size();

	/* CUDA Memory Functions */

	hipMalloc((void**)&d_v, numNeurons * sizeof(float));
	hipMalloc((void**)&d_u, numNeurons * sizeof(float));
	hipMalloc((void**)&d_I, numNeurons * sizeof(float));
	hipMalloc((void**)&d_driven, numNeurons * sizeof(float));
	hipMalloc((void**)&d_cf, numNeurons * sizeof(bool));

	
	hipMalloc((void**)&d_source, numEdges * sizeof(int));
	hipMalloc((void**)&d_target, numEdges * sizeof(int));
	hipMalloc((void**)&d_weight, numEdges * sizeof(float));


	hipMemcpy(d_v, h_v, numNeurons * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_u, h_u, numNeurons * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_I, h_I, numNeurons * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_driven, h_driven, numNeurons * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(d_source, h_source.data(), numEdges * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_target, h_target.data(), numEdges * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_weight, h_weight.data(), numEdges * sizeof(float), hipMemcpyHostToDevice);


	/* Run Simulation */

	for (int t = 0; t < equilizationTime; t++)
	{
		/* Run Timesteps, No Communication */
		NeuronTimestep <<<(numNeurons + numThreads - 1) / numThreads , numThreads >>>(
			numNeurons,
			numExcit,
			d_v,
			d_u,
			d_I,
			d_cf,
			d_driven);
	}

	for (int t = 0; t < transientTime; t++)
	{
		/* Run Timesteps, Communication, No Writing */
		NeuronTimestep << <(numNeurons + numThreads - 1) / numThreads, numThreads >> >(
			numNeurons,
			numExcit,
			d_v,
			d_u,
			d_I,
			d_cf,
			d_driven);

		CommunicationPhase << <(numEdges + numThreads - 1) / numThreads, numThreads >> >(
			numEdges,
			d_cf,
			d_source,
			d_target,
			d_weight,
			d_I);

	}

	for (int t = 0; t < T; t++)
	{
		/* Run Timesteps, Communication, Write Results*/
		NeuronTimestep << <(numNeurons + numThreads - 1) / numThreads, numThreads >> >(
			numNeurons,
			numExcit,
			d_v,
			d_u,
			d_I,
			d_cf,
			d_driven);

		CommunicationPhase<<<(numEdges + numThreads - 1) / numThreads, numThreads>>>(
			numEdges,
			d_cf,
			d_source,
			d_target,
			d_weight,
			d_I);

		hipMemcpy(SpikeTrainYard[t], d_cf, numNeurons * sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(VoltageTrace[t], d_v, numNeurons * sizeof(float), hipMemcpyDeviceToHost);
	}


	/* Analyzing Run */

	std::vector<std::vector<int>> Firings;

	for (int t = 0; t < T; t++)
	{
		for (int n = 0; n < numNeurons; n++)
		{
			if (SpikeTrainYard[t][n] == true)
			{
				std::vector<int> v;
				v.push_back(t);
				v.push_back(n);
				Firings.push_back(v);
			}
		}
	}

	std::cout << "There were " << Firings.size() << " firings." << std::endl;


	/* Clean Up Code */

	hipDeviceReset();

	for (int t = 0; t < T; t++)
	{
		delete[] SpikeTrainYard[t];
		delete[] VoltageTrace[t];
	}

	delete[] h_v; delete[] h_u; delete[] h_I; delete[] h_cf; delete[] SpikeTrainYard; delete[] h_driven;
	delete[] VoltageTrace;
    return 0;
}
